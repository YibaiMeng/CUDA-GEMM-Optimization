#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.hpp"

// GEMM kernel v06.
// Each thread in the block processes THREAD_TILE_SIZE_Y *
// THREAD_TILE_SIZE_X output values. Number of threads BLOCK_TILE_SIZE_Y *
// BLOCK_TILE_SIZE_X / (THREAD_TILE_SIZE_Y * THREAD_TILE_SIZE_X)
template <typename T, size_t BLOCK_TILE_SIZE_X, size_t BLOCK_TILE_SIZE_Y,
          size_t BLOCK_TILE_SIZE_K, size_t WARP_TILE_SIZE_X,
          size_t WARP_TILE_SIZE_Y, size_t THREAD_TILE_SIZE_X,
          size_t THREAD_TILE_SIZE_Y, size_t NUM_THREADS_PER_WARP_X,
          size_t NUM_THREADS_PER_WARP_Y>
__global__ void gemm_v06_vectorized(size_t m, size_t n, size_t k, T alpha,
                                    T const* A, size_t lda, T const* B,
                                    size_t ldb, T beta, T* C, size_t ldc)
{
    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32U);
    constexpr size_t NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0U);
    constexpr size_t NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0U);
    constexpr unsigned int NUM_THREAD_TILES_PER_WARP_X{
        WARP_TILE_SIZE_X / (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X)};
    constexpr unsigned int NUM_THREAD_TILES_PER_WARP_Y{
        WARP_TILE_SIZE_Y / (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y)};
    static_assert(
        WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0U);
    static_assert(
        WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0U);

    constexpr unsigned int NUM_THREADS_X{NUM_WARPS_X * NUM_THREADS_PER_WARP_X};
    constexpr unsigned int NUM_THREADS_Y{NUM_WARPS_Y * NUM_THREADS_PER_WARP_Y};
    // Avoid using blockDim.x * blockDim.y as the number of threads per block.
    // Because it is a runtime constant and the compiler cannot optimize the
    // loop unrolling based on that.
    // Use a compile time constant instead.
    constexpr size_t NUM_THREADS{NUM_THREADS_X * NUM_THREADS_Y};

    // Cache a tile of A and B in shared memory for data reuse.
    __shared__ T
        A_thread_block_tile_transposed[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_Y];
    __shared__ T B_thread_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X];

    // A_vals is cached in the register.
    T A_vals[NUM_THREAD_TILES_PER_WARP_Y][THREAD_TILE_SIZE_Y] = {
        static_cast<T>(0)};
    // B_vals is cached in the register.
    T B_vals[NUM_THREAD_TILES_PER_WARP_X][THREAD_TILE_SIZE_X] = {
        static_cast<T>(0)};

    size_t const thread_linear_idx{threadIdx.y * blockDim.x + threadIdx.x};
    size_t const warp_linear_idx{thread_linear_idx / 32U};
    size_t const warp_row_idx{warp_linear_idx / NUM_WARPS_X};
    size_t const warp_col_idx{warp_linear_idx % NUM_WARPS_X};
    size_t const thread_linear_idx_in_warp{thread_linear_idx % 32U};
    size_t const thread_linear_row_idx_in_warp{thread_linear_idx_in_warp /
                                               NUM_THREADS_PER_WARP_X};
    size_t const thread_linear_col_idx_in_warp{thread_linear_idx_in_warp %
                                               NUM_THREADS_PER_WARP_X};

    // Number of outer loops to perform the sum of inner products.
    // C_thread_block_tile =
    // \sigma_{thread_block_tile_idx=0}^{num_thread_block_tiles-1} A[:,
    // thread_block_tile_idx:BLOCK_TILE_SIZE_K] *
    // B[thread_block_tile_idx:BLOCK_TILE_SIZE_K, :]
    size_t const num_thread_block_tiles{(k + BLOCK_TILE_SIZE_K - 1) /
                                        BLOCK_TILE_SIZE_K};
    // Each thread in the block processes NUM_THREAD_TILES_PER_WARP_Y *
    // NUM_THREAD_TILES_PER_WARP_X * THREAD_TILE_SIZE_Y *
    // THREAD_TILE_SIZE_X output values.
    T C_thread_results[NUM_THREAD_TILES_PER_WARP_Y][NUM_THREAD_TILES_PER_WARP_X]
                      [THREAD_TILE_SIZE_Y][THREAD_TILE_SIZE_X] = {
                          static_cast<T>(0)};

    constexpr size_t NUM_VECTOR_UNITS{sizeof(int4) / sizeof(T)};
    static_assert(sizeof(int4) % sizeof(T) == 0U);
    static_assert(BLOCK_TILE_SIZE_K % NUM_VECTOR_UNITS == 0U);
    static_assert(BLOCK_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_BLOCK_TILE_SIZE_K{BLOCK_TILE_SIZE_K /
                                                  NUM_VECTOR_UNITS};
    static_assert(BLOCK_TILE_SIZE_K % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_BLOCK_TILE_SIZE_X{BLOCK_TILE_SIZE_X /
                                                  NUM_VECTOR_UNITS};
    static_assert(BLOCK_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_X{THREAD_TILE_SIZE_X /
                                                   NUM_VECTOR_UNITS};
    static_assert(THREAD_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_Y{THREAD_TILE_SIZE_Y /
                                                   NUM_VECTOR_UNITS};
    static_assert(THREAD_TILE_SIZE_Y % NUM_VECTOR_UNITS == 0U);

    for (size_t thread_block_tile_idx{0U};
         thread_block_tile_idx < num_thread_block_tiles;
         ++thread_block_tile_idx)
    {
// Load data from A on DRAM to A_thread_block_tile on shared memory.
#pragma unroll
        for (size_t load_idx{0U};
             load_idx < (BLOCK_TILE_SIZE_Y * VECTORIZED_BLOCK_TILE_SIZE_K +
                         NUM_THREADS - 1U) /
                            NUM_THREADS;
             ++load_idx)
        {
            size_t const A_thread_block_tile_row_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) /
                VECTORIZED_BLOCK_TILE_SIZE_K};
            size_t const A_thread_block_tile_col_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) %
                VECTORIZED_BLOCK_TILE_SIZE_K * NUM_VECTOR_UNITS};
            size_t const A_row_idx{blockIdx.y * BLOCK_TILE_SIZE_Y +
                                   A_thread_block_tile_row_idx};
            size_t const A_col_idx{thread_block_tile_idx * BLOCK_TILE_SIZE_K +
                                   A_thread_block_tile_col_idx};

            // These boundary checks might slow down the kernel to some extent.
            // But they guarantee the correctness of the kernel for all
            // different GEMM configurations.
            int4 A_row_vector_vals{0, 0, 0, 0};
            if (A_row_idx < m && A_col_idx < k)
            {
                A_row_vector_vals = *reinterpret_cast<int4 const*>(
                    &A[A_row_idx * lda + A_col_idx]);
            }
            if (A_col_idx + NUM_VECTOR_UNITS > k)
            {
                // Number of invalid elements in the last vector.
                size_t const num_invalid_elements{A_col_idx + NUM_VECTOR_UNITS -
                                                  k};
                // Mask out the invalid elements.
                T* const A_row_vector_vals_ptr{
                    reinterpret_cast<T*>(&A_row_vector_vals)};
                for (size_t i{0U}; i < num_invalid_elements; ++i)
                {
                    A_row_vector_vals_ptr[NUM_VECTOR_UNITS - 1U - i] =
                        static_cast<T>(0);
                }
            }
            // If this is true, the following if can be removed.
            // static_assert(VECTORIZED_BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y %
            // NUM_THREADS ==
            //               0U);
            if (A_thread_block_tile_row_idx < BLOCK_TILE_SIZE_Y &&
                A_thread_block_tile_col_idx < BLOCK_TILE_SIZE_K)
            {
                for (size_t i{0U}; i < NUM_VECTOR_UNITS; ++i)
                {
                    A_thread_block_tile_transposed
                        [A_thread_block_tile_col_idx + i]
                        [A_thread_block_tile_row_idx] =
                            reinterpret_cast<T const*>(&A_row_vector_vals)[i];
                }
            }
        }
// Load data from B on DRAM to B_thread_block_tile on shared memory.
#pragma unroll
        for (size_t load_idx{0U};
             load_idx < (BLOCK_TILE_SIZE_K * VECTORIZED_BLOCK_TILE_SIZE_X +
                         NUM_THREADS - 1U) /
                            NUM_THREADS;
             ++load_idx)
        {
            size_t const B_thread_block_tile_row_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) /
                VECTORIZED_BLOCK_TILE_SIZE_X};
            size_t const B_thread_block_tile_col_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) %
                VECTORIZED_BLOCK_TILE_SIZE_X * NUM_VECTOR_UNITS};
            size_t const B_row_idx{thread_block_tile_idx * BLOCK_TILE_SIZE_K +
                                   B_thread_block_tile_row_idx};
            size_t const B_col_idx{blockIdx.x * BLOCK_TILE_SIZE_X +
                                   B_thread_block_tile_col_idx};

            // These boundary checks might slow down the kernel to some extent.
            // But they guarantee the correctness of the kernel for all
            // different GEMM configurations.
            int4 B_row_vector_vals{0, 0, 0, 0};
            if (B_row_idx < k && B_col_idx < n)
            {
                B_row_vector_vals = *reinterpret_cast<int4 const*>(
                    &B[B_row_idx * ldb + B_col_idx]);
            }
            if (B_col_idx + NUM_VECTOR_UNITS > n)
            {
                // Number of invalid elements in the last vector.
                size_t const num_invalid_elements{B_col_idx + NUM_VECTOR_UNITS -
                                                  n};
                // Mask out the invalid elements.
                T* const B_row_vector_vals_ptr{
                    reinterpret_cast<T*>(&B_row_vector_vals)};
                for (size_t i{0U}; i < num_invalid_elements; ++i)
                {
                    B_row_vector_vals_ptr[NUM_VECTOR_UNITS - 1U - i] =
                        static_cast<T>(0);
                }
            }
            // If this is true, the following if can be removed.
            // static_assert(VECTORIZED_BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K %
            // NUM_THREADS ==
            //               0U);
            if (B_thread_block_tile_row_idx < BLOCK_TILE_SIZE_K &&
                B_thread_block_tile_col_idx < BLOCK_TILE_SIZE_X)
            {
                *reinterpret_cast<int4*>(
                    &B_thread_block_tile[B_thread_block_tile_row_idx]
                                        [B_thread_block_tile_col_idx]) =
                    B_row_vector_vals;
            }
        }
        __syncthreads();

// Perform A[:, thread_block_tile_idx:BLOCK_TILE_SIZE_K] *
// B[thread_block_tile_idx:BLOCK_TILE_SIZE_K, :] where A[:,
// thread_block_tile_idx:BLOCK_TILE_SIZE_K] and
// B[thread_block_tile_idx:BLOCK_TILE_SIZE_K, :] are cached in the
// shared memory as A_thread_block_tile and B_thread_block_tile,
// respectively. This inner product is further decomposed to
// BLOCK_TILE_SIZE_K outer products. A_thread_block_tile *
// B_thread_block_tile = \sigma_{k_i=0}^{BLOCK_TILE_SIZE_K-1}
// A_thread_block_tile[:, k_i] @ B_thread_block_tile[k_i, :] Note that
// both A_thread_block_tile and B_thread_block_tile can be cached in the
// register.
// Can use pragma unroll to unroll these static loops to see if there is a
// performance gain.
#pragma unroll
        for (size_t k_i{0U}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
        {
#pragma unroll
            for (size_t thread_tile_repeat_row_idx{0U};
                 thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_Y;
                 ++thread_tile_repeat_row_idx)
            {
                size_t const A_thread_block_tile_row_idx{
                    warp_row_idx * WARP_TILE_SIZE_Y +
                    thread_tile_repeat_row_idx *
                        (WARP_TILE_SIZE_Y / NUM_THREAD_TILES_PER_WARP_Y) +
                    thread_linear_row_idx_in_warp * THREAD_TILE_SIZE_Y};
                size_t const A_thread_block_tile_col_idx{k_i};
#pragma unroll
                for (size_t thread_tile_y_vector_idx{0U};
                     thread_tile_y_vector_idx < VECTORIZED_THREAD_TILE_SIZE_Y;
                     ++thread_tile_y_vector_idx)
                {
                    *reinterpret_cast<int4*>(
                        &A_vals[thread_tile_repeat_row_idx]
                               [thread_tile_y_vector_idx * NUM_VECTOR_UNITS]) =
                        *reinterpret_cast<int4 const*>(
                            &A_thread_block_tile_transposed
                                [A_thread_block_tile_col_idx]
                                [A_thread_block_tile_row_idx +
                                 thread_tile_y_vector_idx * NUM_VECTOR_UNITS]);
                }
            }
#pragma unroll
            for (size_t thread_tile_repeat_col_idx{0U};
                 thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_X;
                 ++thread_tile_repeat_col_idx)
            {
                size_t const B_thread_block_tile_row_idx{k_i};
                size_t const B_thread_block_tile_col_idx{
                    warp_col_idx * WARP_TILE_SIZE_X +
                    thread_tile_repeat_col_idx *
                        (WARP_TILE_SIZE_X / NUM_THREAD_TILES_PER_WARP_X) +
                    thread_linear_col_idx_in_warp * THREAD_TILE_SIZE_X};
#pragma unroll
                for (size_t thread_tile_x_vector_idx{0U};
                     thread_tile_x_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
                     ++thread_tile_x_vector_idx)
                {
                    *reinterpret_cast<int4*>(
                        &B_vals[thread_tile_repeat_col_idx]
                               [thread_tile_x_vector_idx * NUM_VECTOR_UNITS]) =
                        *reinterpret_cast<int4 const*>(
                            &B_thread_block_tile[B_thread_block_tile_row_idx]
                                                [B_thread_block_tile_col_idx +
                                                 thread_tile_x_vector_idx *
                                                     NUM_VECTOR_UNITS]);
                }
            }

// Compute NUM_THREAD_TILES_PER_WARP_Y * NUM_THREAD_TILES_PER_WARP_X outer
// products.
#pragma unroll
            for (size_t thread_tile_repeat_row_idx{0U};
                 thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_Y;
                 ++thread_tile_repeat_row_idx)
            {
#pragma unroll
                for (size_t thread_tile_repeat_col_idx{0U};
                     thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_X;
                     ++thread_tile_repeat_col_idx)
                {
#pragma unroll
                    for (size_t thread_tile_y_idx{0U};
                         thread_tile_y_idx < THREAD_TILE_SIZE_Y;
                         ++thread_tile_y_idx)
                    {
#pragma unroll
                        for (size_t thread_tile_x_idx{0U};
                             thread_tile_x_idx < THREAD_TILE_SIZE_X;
                             ++thread_tile_x_idx)
                        {
                            C_thread_results[thread_tile_repeat_row_idx]
                                            [thread_tile_repeat_col_idx]
                                            [thread_tile_y_idx]
                                            [thread_tile_x_idx] +=
                                A_vals[thread_tile_repeat_row_idx]
                                      [thread_tile_y_idx] *
                                B_vals[thread_tile_repeat_col_idx]
                                      [thread_tile_x_idx];
                        }
                    }
                }
            }
        }
        // We can use syncwarp now.
        __syncwarp();
    }
    // Need a synchronization before writing the results to DRAM.
    __syncthreads();

// Write the results to DRAM.
#pragma unroll
    for (size_t thread_tile_repeat_row_idx{0U};
         thread_tile_repeat_row_idx < NUM_THREAD_TILES_PER_WARP_Y;
         ++thread_tile_repeat_row_idx)
    {
#pragma unroll
        for (size_t thread_tile_repeat_col_idx{0U};
             thread_tile_repeat_col_idx < NUM_THREAD_TILES_PER_WARP_X;
             ++thread_tile_repeat_col_idx)
        {
#pragma unroll
            for (size_t thread_tile_y_idx{0U};
                 thread_tile_y_idx < THREAD_TILE_SIZE_Y; ++thread_tile_y_idx)
            {
#pragma unroll
                for (size_t thread_tile_x_vector_idx{0U};
                     thread_tile_x_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
                     ++thread_tile_x_vector_idx)
                {
                    size_t const C_row_idx{
                        blockIdx.y * BLOCK_TILE_SIZE_Y +
                        warp_row_idx * WARP_TILE_SIZE_Y +
                        thread_tile_repeat_row_idx *
                            (WARP_TILE_SIZE_Y / NUM_THREAD_TILES_PER_WARP_Y) +
                        thread_linear_row_idx_in_warp * THREAD_TILE_SIZE_Y +
                        thread_tile_y_idx};
                    size_t const C_col_idx{
                        blockIdx.x * BLOCK_TILE_SIZE_X +
                        warp_col_idx * WARP_TILE_SIZE_X +
                        thread_tile_repeat_col_idx *
                            (WARP_TILE_SIZE_X / NUM_THREAD_TILES_PER_WARP_X) +
                        thread_linear_col_idx_in_warp * THREAD_TILE_SIZE_X +
                        thread_tile_x_vector_idx * NUM_VECTOR_UNITS};

                    if (C_row_idx < m && C_col_idx < n)
                    {
                        int4 C_vals{*reinterpret_cast<int4 const*>(
                            &C[C_row_idx * ldc + C_col_idx])};
#pragma unroll
                        for (size_t i{0U}; i < NUM_VECTOR_UNITS; ++i)
                        {
                            reinterpret_cast<T*>(&C_vals)[i] =
                                alpha *
                                    C_thread_results[thread_tile_repeat_row_idx]
                                                    [thread_tile_repeat_col_idx]
                                                    [thread_tile_y_idx]
                                                    [thread_tile_x_vector_idx *
                                                         NUM_VECTOR_UNITS +
                                                     i] +
                                beta * reinterpret_cast<T const*>(&C_vals)[i];
                        }
                        *reinterpret_cast<int4*>(
                            &C[C_row_idx * ldc + C_col_idx]) = C_vals;
                    }
                }
            }
        }
    }
}

template <typename T>
void launch_gemm_kernel_v06_vectorized(size_t m, size_t n, size_t k,
                                       T const* alpha, T const* A, size_t lda,
                                       T const* B, size_t ldb, T const* beta,
                                       T* C, size_t ldc, hipStream_t stream)
{
    // Feel free to play with the block tile sizes.
    // The algorithm correctness should always be guaranteed.
    constexpr unsigned int BLOCK_TILE_SIZE_X{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_Y{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_K{16U};

    constexpr unsigned int WARP_TILE_SIZE_X{32U};
    constexpr unsigned int WARP_TILE_SIZE_Y{64U};
    constexpr unsigned int NUM_WARPS_X{BLOCK_TILE_SIZE_X / WARP_TILE_SIZE_X};
    constexpr unsigned int NUM_WARPS_Y{BLOCK_TILE_SIZE_Y / WARP_TILE_SIZE_Y};
    static_assert(BLOCK_TILE_SIZE_X % WARP_TILE_SIZE_X == 0U);
    static_assert(BLOCK_TILE_SIZE_Y % WARP_TILE_SIZE_Y == 0U);

    constexpr unsigned int THREAD_TILE_SIZE_X{8U};
    constexpr unsigned int THREAD_TILE_SIZE_Y{8U};

    constexpr unsigned int NUM_THREADS_PER_WARP_X{4U};
    constexpr unsigned int NUM_THREADS_PER_WARP_Y{8U};
    static_assert(NUM_THREADS_PER_WARP_X * NUM_THREADS_PER_WARP_Y == 32U);
    static_assert(
        WARP_TILE_SIZE_X % (THREAD_TILE_SIZE_X * NUM_THREADS_PER_WARP_X) == 0U);
    static_assert(
        WARP_TILE_SIZE_Y % (THREAD_TILE_SIZE_Y * NUM_THREADS_PER_WARP_Y) == 0U);

    constexpr unsigned int NUM_THREADS_X{NUM_WARPS_X * NUM_THREADS_PER_WARP_X};
    constexpr unsigned int NUM_THREADS_Y{NUM_WARPS_Y * NUM_THREADS_PER_WARP_Y};

    constexpr unsigned int NUM_THREADS_PER_BLOCK{NUM_THREADS_X * NUM_THREADS_Y};

    dim3 const block_dim{NUM_THREADS_PER_BLOCK, 1U, 1U};
    dim3 const grid_dim{
        (static_cast<unsigned int>(n) + BLOCK_TILE_SIZE_X - 1U) /
            BLOCK_TILE_SIZE_X,
        (static_cast<unsigned int>(m) + BLOCK_TILE_SIZE_Y - 1U) /
            BLOCK_TILE_SIZE_Y,
        1U};
    gemm_v06_vectorized<T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y,
                        BLOCK_TILE_SIZE_K, WARP_TILE_SIZE_X, WARP_TILE_SIZE_Y,
                        THREAD_TILE_SIZE_X, THREAD_TILE_SIZE_Y,
                        NUM_THREADS_PER_WARP_X, NUM_THREADS_PER_WARP_Y>
        <<<grid_dim, block_dim, 0U, stream>>>(m, n, k, *alpha, A, lda, B, ldb,
                                              *beta, C, ldc);
    CHECK_LAST_CUDA_ERROR();
}

// Explicit instantiation.
template void launch_gemm_kernel_v06_vectorized<float>(
    size_t m, size_t n, size_t k, float const* alpha, float const* A,
    size_t lda, float const* B, size_t ldb, float const* beta, float* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v06_vectorized<double>(
    size_t m, size_t n, size_t k, double const* alpha, double const* A,
    size_t lda, double const* B, size_t ldb, double const* beta, double* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v06_vectorized<__half>(
    size_t m, size_t n, size_t k, __half const* alpha, __half const* A,
    size_t lda, __half const* B, size_t ldb, __half const* beta, __half* C,
    size_t ldc, hipStream_t stream);