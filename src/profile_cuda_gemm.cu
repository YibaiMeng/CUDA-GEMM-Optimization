#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include "cuda_gemm.hpp"
#include "profile_utils.cuh"

int main()
{
    print_device_info();

    constexpr size_t num_repeats{1U};
    constexpr size_t num_warmups{1U};

    float const fp32_abs_tol{1.0e-3f};
    double const fp32_rel_tol{0.0e-4f};

    __half const fp16_abs_tol{__float2half(1.0e-2f)};
    double const fp16_rel_tol{2.0e-2f};

    // constexpr size_t m{4096U};
    // constexpr size_t k{4096U};
    // constexpr size_t n{4096U};

    // constexpr size_t m{2048U};
    // constexpr size_t k{2048U};
    // constexpr size_t n{2048U};

    constexpr size_t m{1024U};
    constexpr size_t k{1024U};
    constexpr size_t n{1024U};

    // constexpr size_t m{256U};
    // constexpr size_t k{256U};
    // constexpr size_t n{256U};

    // constexpr size_t m{1372U};
    // constexpr size_t k{1153U};
    // constexpr size_t n{2171U};

    // constexpr size_t lda{m};
    // constexpr size_t ldb{k};
    // constexpr size_t ldc{n};

    constexpr size_t lda{(k + 16U - 1U) / 16U * 16U};
    constexpr size_t ldb{(n + 16U - 1U) / 16U * 16U};
    constexpr size_t ldc{(n + 16U - 1U) / 16U * 16U};

    static_assert(lda >= k);
    static_assert(ldb >= n);
    static_assert(ldc >= n);

    std::cout << "Matrix Size: "
              << "M = " << m << " N = " << n << " K = " << k << std::endl;
    std::cout << "Matrix A: " << m << " x " << k
              << " Leading Dimension Size = " << lda << std::endl;
    std::cout << "Matrix B: " << k << " x " << n
              << " Leading Dimension Size = " << ldb << std::endl;
    std::cout << "Matrix C: " << m << " x " << n
              << " Leading Dimension Size = " << ldc << std::endl;
    std::cout << std::endl;

    // Define all the GEMM kernel launch functions to be profiled.
    std::vector<std::pair<
        std::string,
        std::function<void(size_t, size_t, size_t, float const*, float const*,
                           size_t, float const*, size_t, float const*, float*,
                           size_t, hipStream_t)>>> const
        gemm_kernel_launch_functions{
            // {"Custom GEMM Kernel V00", launch_gemm_kernel_v00<float>},
            // {"Custom GEMM Kernel V01", launch_gemm_kernel_v01<float>},
            // {"Custom GEMM Kernel V02", launch_gemm_kernel_v02<float>},
            // {"Custom GEMM Kernel V02 Vectorized",
            //  launch_gemm_kernel_v02_vectorized<float>},
            // {"Custom GEMM Kernel V03", launch_gemm_kernel_v03<float>},
            // {"Custom GEMM Kernel V03 Vectorized",
            //  launch_gemm_kernel_v03_vectorized<float>},
            // {"Custom GEMM Kernel V04", launch_gemm_kernel_v04<float>},
            // {"Custom GEMM Kernel V04 Vectorized",
            //  launch_gemm_kernel_v04_vectorized<float>},
            // {"Custom GEMM Kernel V05", launch_gemm_kernel_v05<float>},
            // {"Custom GEMM Kernel V05 Vectorized",
            //  launch_gemm_kernel_v05_vectorized<float>},
            {"Custom GEMM Kernel V06", launch_gemm_kernel_v06<float>},
            {"Custom GEMM Kernel V06 Vectorized",
             launch_gemm_kernel_v06_vectorized<float>}};

    for (auto const& gemm_kernel_launch_function : gemm_kernel_launch_functions)
    {
        std::cout << gemm_kernel_launch_function.first << std::endl;
        std::pair<float, float> const gemm_kernel_profile_result{
            profile_gemm<float>(
                m, n, k, lda, ldb, ldc, gemm_kernel_launch_function.second,
                fp32_abs_tol, fp32_rel_tol, num_repeats, num_warmups)};
        std::cout << std::endl;
    }

    std::vector<std::pair<
        std::string,
        std::function<void(size_t, size_t, size_t, __half const*, __half const*,
                           size_t, __half const*, size_t, __half const*,
                           __half*, size_t, hipStream_t)>>> const
        gemm_fp16_kernel_launch_functions{
            // {"Custom GEMM Kernel V00", launch_gemm_kernel_v00<__half>},
            // {"Custom GEMM Kernel V01", launch_gemm_kernel_v01<__half>},
            // {"Custom GEMM Kernel V02", launch_gemm_kernel_v02<__half>},
            // {"Custom GEMM Kernel V02 Vectorized",
            //  launch_gemm_kernel_v02_vectorized<__half>},
            // {"Custom GEMM Kernel V03", launch_gemm_kernel_v03<__half>},
            // {"Custom GEMM Kernel V03 Vectorized",
            //  launch_gemm_kernel_v03_vectorized<__half>},
            // {"Custom GEMM Kernel V04", launch_gemm_kernel_v04<__half>},
            // {"Custom GEMM Kernel V04 Vectorized",
            //  launch_gemm_kernel_v04_vectorized<__half>},
            // {"Custom GEMM Kernel V05", launch_gemm_kernel_v05<__half>},
            // {"Custom GEMM Kernel V05 Vectorized",
            //  launch_gemm_kernel_v05_vectorized<__half>},
            // {"Custom GEMM Kernel V06", launch_gemm_kernel_v06<__half>},
            // {"Custom GEMM Kernel V06 Vectorized",
            //  launch_gemm_kernel_v06_vectorized<__half>},
            {"Custom GEMM Kernel V07", launch_gemm_kernel_v07<__half>},
            {"Custom GEMM Kernel V07 Vectorized",
             launch_gemm_kernel_v07_vectorized<__half>}};

    for (auto const& gemm_fp16_kernel_launch_function :
         gemm_fp16_kernel_launch_functions)
    {
        std::cout << gemm_fp16_kernel_launch_function.first << std::endl;
        std::pair<__half, __half> const gemm_kernel_profile_result{
            profile_gemm<__half>(
                m, n, k, lda, ldb, ldc, gemm_fp16_kernel_launch_function.second,
                fp16_abs_tol, fp16_rel_tol, num_repeats, num_warmups)};
        std::cout << std::endl;
    }

    return 0;
}