#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <functional>
#include <iostream>
#include <random>

#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.hpp"

#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

template <typename T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, size_t num_repeats = 100,
                          size_t num_warmups = 100)
{
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (size_t i{0}; i < num_warmups; ++i)
    {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (size_t i{0}; i < num_repeats; ++i)
    {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / num_repeats};

    return latency;
}

#define CHECK_CUBLASS_ERROR(val) check_cublass((val), #val, __FILE__, __LINE__)
void check_cublass(hipblasStatus_t err, const char* const func,
                   const char* const file, const int line)
{
    if (err != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "cuBLAS Error at: " << file << ":" << line << std::endl;
        std::cerr << cublasGetStatusString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

// Determine CUDA data type from type.
template <typename T,
          typename std::enable_if<std::is_same<T, float>::value ||
                                      std::is_same<T, double>::value ||
                                      std::is_same<T, __half>::value,
                                  bool>::type = true>
constexpr hipDataType cuda_data_type_trait()
{
    if (std::is_same<T, float>::value)
    {
        return HIP_R_32F;
    }
    else if (std::is_same<T, double>::value)
    {
        return HIP_R_64F;
    }
    else if (std::is_same<T, __half>::value)
    {
        return HIP_R_16F;
    }
    else
    {
        throw std::runtime_error("Unsupported data type.");
    }
}

template <typename T,
          typename std::enable_if<std::is_same<T, float>::value ||
                                      std::is_same<T, double>::value ||
                                      std::is_same<T, __half>::value,
                                  bool>::type = true>
void launch_gemm_cublas(size_t m, size_t n, size_t k, T const* alpha,
                        T const* A, size_t lda, T const* B, size_t ldb,
                        T const* beta, T* C, size_t ldc, hipblasHandle_t handle)
{
    // Non-TensorCore algorithm?
    constexpr hipblasGemmAlgo_t algo{HIPBLAS_GEMM_DEFAULT};
    constexpr hipDataType data_type{cuda_data_type_trait<T>()};
    // All the matrix are in row-major order.
    // https://docs.nvidia.com/cuda/cublas/#cublasgemmex
    // A: m x k row-major -> A: k x m column-major non-transposed
    // B: k x n row-major -> B: n x k column-major non-transposed
    // C: m x n row-major -> C: n x m column-major non-transposed
    // Thus, without padding, the leading dimension of the matrix in row-major
    // order is the number of columns, i.e., k for A, n for B, and n for C.
    // Row-major order: C = AB + C
    // Column-major order: C = BA + C
    // The cuBLAS API requires the leading dimension of the matrix in
    // column-major order. This API call looks non-intuitive, but it is correct.
    CHECK_CUBLASS_ERROR(hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, alpha, B, data_type, ldb, A,
        data_type, lda, beta, C, data_type, ldc, data_type, algo));
}

template <typename T>
bool all_close(T const* C, T const* C_ref, size_t m, size_t n, size_t ldc,
               T abs_tol)
{
    bool status{true};
    for (size_t i{0U}; i < m; ++i)
    {
        for (size_t j{0U}; j < n; ++j)
        {
            double const C_val{static_cast<double>(C[i * ldc + j])};
            double const C_ref_val{static_cast<double>(C_ref[i * ldc + j])};
            double const diff{C_val - C_ref_val};
            double const diff_val{std::abs(diff)};
            if (diff_val > static_cast<double>(abs_tol))
            {
                std::cout << "C[" << i << ", " << j << "] = " << C_val
                          << " C_ref[" << i << ", " << j << "] = " << C_ref_val
                          << std::endl;
                status = false;
                return status;
            }
        }
    }
    return status;
}

void print_device_info()
{
    int device_id{0};
    hipGetDevice(&device_id);
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device_id);
    std::cout << "Device Name: " << device_prop.name << std::endl;
    float const memory_size{static_cast<float>(device_prop.totalGlobalMem) /
                            (1 << 30)};
    std::cout << "Memory Size: " << memory_size << " GB" << std::endl;
    float const peak_bandwidth{
        static_cast<float>(2.0f * device_prop.memoryClockRate *
                           (device_prop.memoryBusWidth / 8) / 1.0e6)};
    std::cout << "Peak Bandwitdh: " << peak_bandwidth << " GB/s" << std::endl;
    std::cout << std::endl;
}

template <typename T>
float compute_effective_bandwidth(size_t m, size_t n, size_t k, float latency)
{
    return ((m * k + k * n + m * n) * sizeof(T)) / (latency * 1e-3) / 1e9;
}

float compute_effective_tflops(size_t m, size_t n, size_t k, float latency)
{
    return (2.0 * m * k * n) / (latency * 1e-3) / 1e12;
}

template <typename T,
          typename std::enable_if<std::is_same<T, float>::value ||
                                      std::is_same<T, double>::value ||
                                      std::is_same<T, __half>::value,
                                  bool>::type = true>
void random_initialize_matrix(T* A, size_t m, size_t n, size_t lda,
                              unsigned int seed = 0U)
{
    std::mt19937 gen(seed);
    std::uniform_real_distribution<double> dis(0.0, 1.0);
    auto const rand = [&dis, &gen]() { return dis(gen); };
    for (size_t i{0U}; i < m; ++i)
    {
        for (size_t j{0U}; j < n; ++j)
        {
            A[i * lda + j] = static_cast<T>(rand());
        }
    }
}

void print_performance_result(size_t m, size_t n, size_t k, float latency)
{
    float const effective_bandwidth{
        compute_effective_bandwidth<float>(m, n, k, latency)};
    float const effective_tflops{compute_effective_tflops(m, n, k, latency)};

    std::cout << "Latency: " << latency << " ms" << std::endl;
    std::cout << "Effective Bandwidth: " << effective_bandwidth << " GB/s"
              << std::endl;
    std::cout << "Effective TFLOPS: " << effective_tflops << " TFLOPS"
              << std::endl;
}

template <typename T,
          typename std::enable_if<std::is_same<T, float>::value ||
                                      std::is_same<T, double>::value ||
                                      std::is_same<T, __half>::value,
                                  bool>::type = true>
std::pair<float, float> profile_gemm(
    size_t m, size_t n, size_t k, size_t lda, size_t ldb, size_t ldc,
    std::function<void(size_t, size_t, size_t, T const*, T const*, size_t,
                       T const*, size_t, T const*, T*, size_t, hipStream_t)>
        gemm_kernel_launch_function,
    size_t num_repeats = 10, size_t num_warmups = 10, unsigned int seed = 0U)
{
    T const alpha{static_cast<T>(1.0)};
    T const beta{static_cast<T>(0.0)};

    T const abs_tol{static_cast<T>(1.0e-2)};

    // Create CUDA stream.
    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    // Allocate memory on host.
    T* A_host{nullptr};
    T* B_host{nullptr};
    T* C_host{nullptr};
    T* C_host_ref{nullptr};
    T* C_host_from_device{nullptr};
    CHECK_CUDA_ERROR(hipHostMalloc(&A_host, m * lda * sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&B_host, k * ldb * sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&C_host, m * ldc * sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&C_host_ref, m * ldc * sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&C_host_from_device, m * ldc * sizeof(T)));

    // Initialize matrix A and B.
    random_initialize_matrix(A_host, m, k, lda);
    random_initialize_matrix(B_host, k, n, ldb);
    random_initialize_matrix(C_host, m, n, ldc);

    // Allocate memory on device.
    T* A_device{nullptr};
    T* B_device{nullptr};
    T* C_device{nullptr};
    CHECK_CUDA_ERROR(hipMalloc(&A_device, m * lda * sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&B_device, k * ldb * sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&C_device, m * ldc * sizeof(T)));

    // Copy matrix A and B from host to device.
    CHECK_CUDA_ERROR(hipMemcpy(A_device, A_host, m * lda * sizeof(T),
                                hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(B_device, B_host, k * ldb * sizeof(T),
                                hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(C_device, C_host, m * ldc * sizeof(T),
                                hipMemcpyHostToDevice));

    // Create cuBLAS handle.
    hipblasHandle_t handle;
    CHECK_CUBLASS_ERROR(hipblasCreate(&handle));
    CHECK_CUBLASS_ERROR(hipblasSetStream(handle, stream));

    // Compute reference output.
    launch_gemm_cublas<T>(m, n, k, &alpha, A_device, lda, B_device, ldb, &beta,
                          C_device, ldc, handle);
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    // Copy matrix C from device to host.
    CHECK_CUDA_ERROR(hipMemcpy(C_host_ref, C_device, m * ldc * sizeof(T),
                                hipMemcpyDeviceToHost));

    // Launch cuBLAS GEMM.
    float const latency_cublas{measure_performance<T>(
        [&](hipStream_t stream)
        {
            launch_gemm_cublas<T>(m, n, k, &alpha, A_device, lda, B_device, ldb,
                                  &beta, C_device, ldc, handle);
            return 0.0f;
        },
        stream, num_repeats, num_warmups)};

    // Launch CUDA GEMM.
    CHECK_CUDA_ERROR(hipMemcpy(C_device, C_host, m * ldc * sizeof(T),
                                hipMemcpyHostToDevice));
    // Verify the correctness of CUDA GEMM.
    gemm_kernel_launch_function(m, n, k, &alpha, A_device, lda, B_device, ldb,
                                &beta, C_device, ldc, stream);
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    CHECK_CUDA_ERROR(hipMemcpy(C_host_from_device, C_device,
                                m * ldc * sizeof(T), hipMemcpyDeviceToHost));
    assert(all_close<T>(C_host_from_device, C_host_ref, m, n, ldc, abs_tol));

    float const latency_cuda_gemm{measure_performance<T>(
        [&](hipStream_t stream)
        {
            gemm_kernel_launch_function(m, n, k, &alpha, A_device, lda,
                                        B_device, ldb, &beta, C_device, ldc,
                                        stream);
            return 0.0f;
        },
        stream, num_repeats, num_warmups)};

    // Release resources.
    CHECK_CUDA_ERROR(hipFree(A_device));
    CHECK_CUDA_ERROR(hipFree(B_device));
    CHECK_CUDA_ERROR(hipFree(C_device));
    CHECK_CUDA_ERROR(hipHostFree(A_host));
    CHECK_CUDA_ERROR(hipHostFree(B_host));
    CHECK_CUDA_ERROR(hipHostFree(C_host));
    CHECK_CUDA_ERROR(hipHostFree(C_host_ref));
    CHECK_CUDA_ERROR(hipHostFree(C_host_from_device));
    CHECK_CUBLASS_ERROR(hipblasDestroy(handle));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));

    std::cout << "cuBLAS GEMM Kernel Performance" << std::endl;
    print_performance_result(m, n, k, latency_cublas);
    std::cout << "Custom GEMM Kernel Performance" << std::endl;
    print_performance_result(m, n, k, latency_cuda_gemm);
    std::cout << "Custom GEMM VS cuBLAS GEMM Performance: "
              << latency_cublas / latency_cuda_gemm * 100.0f << "%"
              << std::endl;

    return std::pair<float, float>{latency_cublas, latency_cuda_gemm};
}

int main()
{
    print_device_info();

    constexpr size_t num_repeats{20U};
    constexpr size_t num_warmups{20U};

    // constexpr size_t m{4096U};
    // constexpr size_t k{4096U};
    // constexpr size_t n{4096U};

    // constexpr size_t m{2048U};
    // constexpr size_t k{2048U};
    // constexpr size_t n{2048U};

    constexpr size_t m{1372U};
    constexpr size_t k{1153U};
    constexpr size_t n{2171U};

    // constexpr size_t lda{m};
    // constexpr size_t ldb{k};
    // constexpr size_t ldc{n};

    constexpr size_t lda{(k + 16U - 1U) / 16U * 16U};
    constexpr size_t ldb{(n + 16U - 1U) / 16U * 16U};
    constexpr size_t ldc{(n + 16U - 1U) / 16U * 16U};

    static_assert(lda >= k);
    static_assert(ldb >= n);
    static_assert(ldc >= n);

    std::cout << "Matrix Size: "
              << "M = " << m << " N = " << n << " K = " << k << std::endl;
    std::cout << "Matrix A: " << m << " x " << k
              << " Leading Dimension Size = " << lda << std::endl;
    std::cout << "Matrix B: " << k << " x " << n
              << " Leading Dimension Size = " << lda << std::endl;
    std::cout << "Matrix C: " << m << " x " << n
              << " Leading Dimension Size = " << lda << std::endl;
    std::cout << std::endl;

    // Define all the GEMM kernel launch functions to be profiled.
    std::vector<std::pair<
        std::string,
        std::function<void(size_t, size_t, size_t, float const*, float const*,
                           size_t, float const*, size_t, float const*, float*,
                           size_t, hipStream_t)>>> const
        gemm_kernel_launch_functions{
            {"Custom GEMM Kernel V00", launch_gemm_kernel_v00<float>},
            {"Custom GEMM Kernel V01", launch_gemm_kernel_v01<float>},
            {"Custom GEMM Kernel V02", launch_gemm_kernel_v02<float>},
            {"Custom GEMM Kernel V02 Vectorized",
             launch_gemm_kernel_v02_vectorized<float>},
            {"Custom GEMM Kernel V03", launch_gemm_kernel_v03<float>},
            {"Custom GEMM Kernel V03 Vectorized",
             launch_gemm_kernel_v03_vectorized<float>},
            {"Custom GEMM Kernel V04", launch_gemm_kernel_v04<float>},
            {"Custom GEMM Kernel V04 Vectorized",
             launch_gemm_kernel_v04_vectorized<float>},
            {"Custom GEMM Kernel V05", launch_gemm_kernel_v05<float>},
            {"Custom GEMM Kernel V05 Vectorized",
             launch_gemm_kernel_v05_vectorized<float>}};

    for (auto const& gemm_kernel_launch_function : gemm_kernel_launch_functions)
    {
        std::cout << gemm_kernel_launch_function.first << std::endl;
        std::pair<float, float> const gemm_kernel_profile_result{
            profile_gemm<float>(m, n, k, lda, ldb, ldc,
                                gemm_kernel_launch_function.second, num_repeats,
                                num_warmups)};
        std::cout << std::endl;
    }

    return 0;
}