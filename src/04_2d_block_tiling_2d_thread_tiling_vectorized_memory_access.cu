#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include "cuda_gemm.hpp"
#include "cuda_gemm_utils.hpp"

// GEMM kernel v04.
// Coalesced read and write from global memory.
template <typename T, size_t BLOCK_TILE_SIZE_X, size_t BLOCK_TILE_SIZE_Y,
          size_t BLOCK_TILE_SIZE_K, size_t THREAD_TILE_SIZE_X,
          size_t THREAD_TILE_SIZE_Y>
__global__ void gemm_v04_vectorized(size_t m, size_t n, size_t k, T alpha,
                                    T const* A, size_t lda, T const* B,
                                    size_t ldb, T beta, T* C, size_t ldc)
{
    // Avoid using blockDim.x * blockDim.y as the number of threads per block.
    // Because it is a runtime constant and the compiler cannot optimize the
    // loop unrolling based on that.
    // Use a compile time constant instead.
    constexpr size_t NUM_THREADS{BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y /
                                 (THREAD_TILE_SIZE_X * THREAD_TILE_SIZE_Y)};
    size_t const thread_linear_idx{threadIdx.y * blockDim.x + threadIdx.x};

    // Cache a tile of A and B in shared memory for data reuse.
    __shared__ T A_thread_block_tile[BLOCK_TILE_SIZE_Y][BLOCK_TILE_SIZE_K];
    __shared__ T B_thread_block_tile[BLOCK_TILE_SIZE_K][BLOCK_TILE_SIZE_X];

    size_t const num_thread_block_tiles{(k + BLOCK_TILE_SIZE_K - 1) /
                                        BLOCK_TILE_SIZE_K};

    // Each thread in the block processes BLOCK_TILE_SIZE_Y output values.
    // Specifically, these values corresponds to
    // C[blockIdx.y * BLOCK_TILE_SIZE_Y + threadIdx.x / BLOCK_TILE_SIZE_X *
    // THREAD_TILE_SIZE_Y : blockIdx.y * BLOCK_TILE_SIZE_Y + (threadIdx.x /
    // BLOCK_TILE_SIZE_X + 1) * THREAD_TILE_SIZE_Y][blockIdx.x *
    // BLOCK_TILE_SIZE_X + threadIdx.x % BLOCK_TILE_SIZE_X *
    // THREAD_TILE_SIZE_X : blockIdx.x * BLOCK_TILE_SIZE_X + (threadIdx.x %
    // BLOCK_TILE_SIZE_X + 1) * THREAD_TILE_SIZE_X]
    T C_thread_results[THREAD_TILE_SIZE_Y][THREAD_TILE_SIZE_X] = {
        static_cast<T>(0)};
    // A_vals is cached in the register.
    T A_vals[THREAD_TILE_SIZE_Y] = {static_cast<T>(0)};
    // B_vals is cached in the register.
    T B_vals[THREAD_TILE_SIZE_X] = {static_cast<T>(0)};

    constexpr size_t NUM_VECTOR_UNITS{sizeof(int4) / sizeof(T)};
    static_assert(sizeof(int4) % sizeof(T) == 0U);
    static_assert(BLOCK_TILE_SIZE_K % NUM_VECTOR_UNITS == 0U);
    static_assert(BLOCK_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_BLOCK_TILE_SIZE_K{BLOCK_TILE_SIZE_K /
                                                  NUM_VECTOR_UNITS};
    static_assert(BLOCK_TILE_SIZE_K % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_BLOCK_TILE_SIZE_X{BLOCK_TILE_SIZE_X /
                                                  NUM_VECTOR_UNITS};
    static_assert(BLOCK_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);
    constexpr size_t VECTORIZED_THREAD_TILE_SIZE_X{THREAD_TILE_SIZE_X /
                                                   NUM_VECTOR_UNITS};
    static_assert(THREAD_TILE_SIZE_X % NUM_VECTOR_UNITS == 0U);

    for (size_t thread_block_tile_idx{0U};
         thread_block_tile_idx < num_thread_block_tiles;
         ++thread_block_tile_idx)
    {
// Load data from A on DRAM to A_thread_block_tile on shared memory.
#pragma unroll
        for (size_t load_idx{0U};
             load_idx < (BLOCK_TILE_SIZE_Y * VECTORIZED_BLOCK_TILE_SIZE_K +
                         NUM_THREADS - 1U) /
                            NUM_THREADS;
             ++load_idx)
        {
            size_t const A_thread_block_tile_row_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) /
                VECTORIZED_BLOCK_TILE_SIZE_K};
            size_t const A_thread_block_tile_col_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) %
                VECTORIZED_BLOCK_TILE_SIZE_K * NUM_VECTOR_UNITS};
            size_t const A_row_idx{blockIdx.y * BLOCK_TILE_SIZE_Y +
                                   A_thread_block_tile_row_idx};
            size_t const A_col_idx{thread_block_tile_idx * BLOCK_TILE_SIZE_K +
                                   A_thread_block_tile_col_idx};

            // These boundary checks might slow down the kernel to some extent.
            // But they guarantee the correctness of the kernel for all
            // different GEMM configurations.
            int4 A_row_vector_vals{0, 0, 0, 0};
            if (A_row_idx < m && A_col_idx < k)
            {
                A_row_vector_vals = *reinterpret_cast<int4 const*>(
                    &A[A_row_idx * lda + A_col_idx]);
            }
            if (A_col_idx + NUM_VECTOR_UNITS > k)
            {
                // Number of invalid elements in the last vector.
                size_t const num_invalid_elements{A_col_idx + NUM_VECTOR_UNITS -
                                                  k};
                // Mask out the invalid elements.
                T* const A_row_vector_vals_ptr{
                    reinterpret_cast<T*>(&A_row_vector_vals)};
                for (size_t i{0U}; i < num_invalid_elements; ++i)
                {
                    A_row_vector_vals_ptr[NUM_VECTOR_UNITS - 1U - i] =
                        static_cast<T>(0);
                }
            }
            // If this is true, the following if can be removed.
            // static_assert(VECTORIZED_BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y %
            // NUM_THREADS ==
            //               0U);
            if (A_thread_block_tile_row_idx < BLOCK_TILE_SIZE_Y &&
                A_thread_block_tile_col_idx < BLOCK_TILE_SIZE_K)
            {
                *reinterpret_cast<int4*>(
                    &A_thread_block_tile[A_thread_block_tile_row_idx]
                                        [A_thread_block_tile_col_idx]) =
                    A_row_vector_vals;
            }
        }
// Load data from B on DRAM to B_thread_block_tile on shared memory.
#pragma unroll
        for (size_t load_idx{0U};
             load_idx < (BLOCK_TILE_SIZE_K * VECTORIZED_BLOCK_TILE_SIZE_X +
                         NUM_THREADS - 1U) /
                            NUM_THREADS;
             ++load_idx)
        {
            size_t const B_thread_block_tile_row_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) /
                VECTORIZED_BLOCK_TILE_SIZE_X};
            size_t const B_thread_block_tile_col_idx{
                (thread_linear_idx + load_idx * NUM_THREADS) %
                VECTORIZED_BLOCK_TILE_SIZE_X * NUM_VECTOR_UNITS};
            size_t const B_row_idx{thread_block_tile_idx * BLOCK_TILE_SIZE_K +
                                   B_thread_block_tile_row_idx};
            size_t const B_col_idx{blockIdx.x * BLOCK_TILE_SIZE_X +
                                   B_thread_block_tile_col_idx};

            // These boundary checks might slow down the kernel to some extent.
            // But they guarantee the correctness of the kernel for all
            // different GEMM configurations.
            int4 B_row_vector_vals{0, 0, 0, 0};
            if (B_row_idx < k && B_col_idx < n)
            {
                B_row_vector_vals = *reinterpret_cast<int4 const*>(
                    &B[B_row_idx * ldb + B_col_idx]);
            }
            if (B_col_idx + NUM_VECTOR_UNITS > n)
            {
                // Number of invalid elements in the last vector.
                size_t const num_invalid_elements{B_col_idx + NUM_VECTOR_UNITS -
                                                  n};
                // Mask out the invalid elements.
                T* const B_row_vector_vals_ptr{
                    reinterpret_cast<T*>(&B_row_vector_vals)};
                for (size_t i{0U}; i < num_invalid_elements; ++i)
                {
                    B_row_vector_vals_ptr[NUM_VECTOR_UNITS - 1U - i] =
                        static_cast<T>(0);
                }
            }
            // If this is true, the following if can be removed.
            // static_assert(VECTORIZED_BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K %
            // NUM_THREADS ==
            //               0U);
            if (B_thread_block_tile_row_idx < BLOCK_TILE_SIZE_K &&
                B_thread_block_tile_col_idx < BLOCK_TILE_SIZE_X)
            {
                *reinterpret_cast<int4*>(
                    &B_thread_block_tile[B_thread_block_tile_row_idx]
                                        [B_thread_block_tile_col_idx]) =
                    B_row_vector_vals;
            }
        }
        __syncthreads();

#pragma unroll
        for (size_t k_i{0U}; k_i < BLOCK_TILE_SIZE_K; ++k_i)
        {
            size_t const A_thread_block_tile_row_idx{
                thread_linear_idx / (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                THREAD_TILE_SIZE_Y};
            size_t const A_thread_block_tile_col_idx{k_i};

#pragma unroll
            for (size_t thread_tile_row_idx{0U};
                 thread_tile_row_idx < THREAD_TILE_SIZE_Y;
                 ++thread_tile_row_idx)
            {
                // There will be shared memory bank conflicts accessing the
                // values from A_thread_block_tile. We can do it better by
                // transposing the A_thread_block_tile when we load the data
                // from DRAM.
                A_vals[thread_tile_row_idx] =
                    A_thread_block_tile[A_thread_block_tile_row_idx +
                                        thread_tile_row_idx]
                                       [A_thread_block_tile_col_idx];
            }

            size_t const B_thread_block_tile_row_idx{k_i};
            size_t const B_thread_block_tile_col_idx{
                thread_linear_idx % (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                THREAD_TILE_SIZE_X};
// Although the read from A_thread_block_tile cannot be vectorized, the read
// from B_thread_block_tile can be vectorized.
#pragma unroll
            for (size_t thread_tile_col_vector_idx{0U};
                 thread_tile_col_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
                 ++thread_tile_col_vector_idx)
            {
                *reinterpret_cast<int4*>(
                    &B_vals[thread_tile_col_vector_idx * NUM_VECTOR_UNITS]) =
                    *reinterpret_cast<int4 const*>(
                        &B_thread_block_tile[B_thread_block_tile_row_idx]
                                            [B_thread_block_tile_col_idx +
                                             thread_tile_col_vector_idx *
                                                 NUM_VECTOR_UNITS]);
            }

            for (size_t thread_tile_row_idx{0U};
                 thread_tile_row_idx < THREAD_TILE_SIZE_Y;
                 ++thread_tile_row_idx)
            {
                for (size_t thread_tile_col_idx{0U};
                     thread_tile_col_idx < THREAD_TILE_SIZE_X;
                     ++thread_tile_col_idx)
                {
                    C_thread_results[thread_tile_row_idx]
                                    [thread_tile_col_idx] +=
                        A_vals[thread_tile_row_idx] *
                        B_vals[thread_tile_col_idx];
                }
            }
        }
        __syncthreads();
    }

    // Vectorized writing the results to DRAM.
    for (size_t thread_tile_row_idx{0U};
         thread_tile_row_idx < THREAD_TILE_SIZE_Y; ++thread_tile_row_idx)
    {
        for (size_t thread_tile_col_vector_idx{0U};
             thread_tile_col_vector_idx < VECTORIZED_THREAD_TILE_SIZE_X;
             ++thread_tile_col_vector_idx)
        {
            size_t const C_row_idx{
                blockIdx.y * BLOCK_TILE_SIZE_Y +
                thread_linear_idx / (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                    THREAD_TILE_SIZE_Y +
                thread_tile_row_idx};
            size_t const C_col_idx{
                blockIdx.x * BLOCK_TILE_SIZE_X +
                thread_linear_idx % (BLOCK_TILE_SIZE_X / THREAD_TILE_SIZE_X) *
                    THREAD_TILE_SIZE_X +
                thread_tile_col_vector_idx * NUM_VECTOR_UNITS};
            // Vectorized read from C.
            int4 C_row_vector_vals{*reinterpret_cast<int4 const*>(
                &C[C_row_idx * ldc + C_col_idx])};
            // Vectorized read from C_thread_results.
            int4 const C_thread_results_row_vector_vals{
                *reinterpret_cast<int4 const*>(
                    &C_thread_results[thread_tile_row_idx]
                                     [thread_tile_col_vector_idx *
                                      NUM_VECTOR_UNITS])};
            // Update the values in C_row_vector_vals
            for (size_t i{0U}; i < NUM_VECTOR_UNITS; ++i)
            {
                reinterpret_cast<T*>(&C_row_vector_vals)[i] =
                    alpha * reinterpret_cast<T const*>(
                                &C_thread_results_row_vector_vals)[i] +
                    beta * reinterpret_cast<T const*>(&C_row_vector_vals)[i];
            }
            // Vectorized write to C.
            if (C_row_idx < m && C_col_idx < n)
            {
                // No need to mask out the out-of-bound invalid elements,
                // because the row of C matrix is 32-byte aligned.
                *reinterpret_cast<int4*>(&C[C_row_idx * ldc + C_col_idx]) =
                    C_row_vector_vals;
            }
        }
    }
}

template <typename T>
void launch_gemm_kernel_v04_vectorized(size_t m, size_t n, size_t k,
                                       T const* alpha, T const* A, size_t lda,
                                       T const* B, size_t ldb, T const* beta,
                                       T* C, size_t ldc, hipStream_t stream)
{
    // Feel free to play with the block tile sizes.
    // The algorithm correctness should always be guaranteed.
    constexpr unsigned int BLOCK_TILE_SIZE_X{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_Y{128U};
    constexpr unsigned int BLOCK_TILE_SIZE_K{16U};
    // Each thread computes THREAD_TILE_SIZE_X * THREAD_TILE_SIZE_Y values of C.
    constexpr unsigned int THREAD_TILE_SIZE_X{8U};
    constexpr unsigned int THREAD_TILE_SIZE_Y{8U};
    constexpr unsigned int NUM_THREADS_PER_BLOCK{
        BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_Y /
        (THREAD_TILE_SIZE_X * THREAD_TILE_SIZE_Y)};
    static_assert(BLOCK_TILE_SIZE_X % THREAD_TILE_SIZE_X == 0U);
    static_assert(BLOCK_TILE_SIZE_Y % THREAD_TILE_SIZE_Y == 0U);
    static_assert(NUM_THREADS_PER_BLOCK % BLOCK_TILE_SIZE_K == 0U);
    static_assert(NUM_THREADS_PER_BLOCK % BLOCK_TILE_SIZE_X == 0U);
    static_assert(
        BLOCK_TILE_SIZE_X * BLOCK_TILE_SIZE_K % NUM_THREADS_PER_BLOCK == 0U);
    static_assert(
        BLOCK_TILE_SIZE_K * BLOCK_TILE_SIZE_Y % NUM_THREADS_PER_BLOCK == 0U);
    dim3 const block_dim{NUM_THREADS_PER_BLOCK, 1U, 1U};
    dim3 const grid_dim{
        (static_cast<unsigned int>(n) + BLOCK_TILE_SIZE_X - 1U) /
            BLOCK_TILE_SIZE_X,
        (static_cast<unsigned int>(m) + BLOCK_TILE_SIZE_Y - 1U) /
            BLOCK_TILE_SIZE_Y,
        1U};
    gemm_v04_vectorized<T, BLOCK_TILE_SIZE_X, BLOCK_TILE_SIZE_Y,
                        BLOCK_TILE_SIZE_K, THREAD_TILE_SIZE_X,
                        THREAD_TILE_SIZE_Y>
        <<<grid_dim, block_dim, 0U, stream>>>(m, n, k, *alpha, A, lda, B, ldb,
                                              *beta, C, ldc);
    CHECK_LAST_CUDA_ERROR();
}

// Explicit instantiation.
template void launch_gemm_kernel_v04_vectorized<float>(
    size_t m, size_t n, size_t k, float const* alpha, float const* A,
    size_t lda, float const* B, size_t ldb, float const* beta, float* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v04_vectorized<double>(
    size_t m, size_t n, size_t k, double const* alpha, double const* A,
    size_t lda, double const* B, size_t ldb, double const* beta, double* C,
    size_t ldc, hipStream_t stream);
template void launch_gemm_kernel_v04_vectorized<__half>(
    size_t m, size_t n, size_t k, __half const* alpha, __half const* A,
    size_t lda, __half const* B, size_t ldb, __half const* beta, __half* C,
    size_t ldc, hipStream_t stream);